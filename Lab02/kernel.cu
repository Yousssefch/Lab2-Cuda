﻿
#include "hip/hip_runtime.h"

#ifndef __CUDACC__ 
#define __CUDACC__
#endif
#include <hip/device_functions.h>
#include <stdio.h>
#include <iostream>
#include <math.h>

using namespace std;




__global__ void neighboredPairSum(int* input, int n, int step) {
    int tid = threadIdx.x; // Thread ID in the block
    int offset = 1 << step; // Calculate the offset for each step
    if (tid % (2 * offset) == 0 && (tid + offset) < n) {
        input[tid] += input[tid + offset];
    }
    __syncthreads(); // Thread synchronization
}



void displayList(int* ptr, int size) {
    cout << "{ ";

    //print list
    for (auto i = 0; i < size; i++)
    {
        cout << ptr[i] << ", ";
    }

    cout << "}";
}

int main()
{
    //init variables
    int N = 10;
    int  *input_host, *input_gpu;
    int n_host, ste_host;

    //alocate space in host
    input_host = (int*)malloc(N * sizeof(int));


    //allocate space in GPU
    hipMalloc(&input_gpu, N * sizeof(int));

    //attribute concecutive values to input host
    for (int i = 0; i < N; i++) {
        input_host[i] = i;
    }

    //print the initial list
    cout << "The initial list is: " << endl;
    displayList(input_host, N);


    //transfer data from host to gpu
    hipMemcpy(input_gpu, input_host, N * sizeof(int), hipMemcpyHostToDevice);

    // Configure kernel execution
    int threads_per_block = 128;
    int num_blocks = (N + threads_per_block - 1) / threads_per_block;

    // Perform neighbored pair sum reduction
    for (int j = 0; j < N; j++) {
        neighboredPairSum <<< num_blocks, threads_per_block >>> (input_gpu, N, j);
    }

    // Copy result back to host
    hipMemcpy(input_host, input_gpu, N * sizeof(int), hipMemcpyDeviceToHost);

    //print the result list
    cout << "The result list is: " << endl;
    displayList(input_host, N);

    // Clean up
    hipFree(input_gpu);
    free(input_host);


    return 0;
}



